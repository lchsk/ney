#include "hip/hip_runtime.h"
#include <iostream>
#include "../../include/ney/ney.hpp"

//! Computes Y = a * X + Y, where X, Y - vectors, a - scalar

int main (int argc, char** argv)
{
    std::cout << "\nSaxpy: Y = a * X + Y\n\n";

    ney::config.target = ney::GPU;
    ney::config.print();

    // start clock

    ney::clock::get().start().precision(7);

    ney::status s;
    ney::vector<int> X = ney::new_vector().size(10);
    ney::vector<int> Y = ney::new_vector().size(10);
    int a = 2;

    s = ney::random<int>(X).min(0).max(10);
    s = ney::random<int>(Y).min(0).max(10);

    std::cout << "scalar a: " << a << "\n";
    std::cout << "X: \n" << X << "\n";
    std::cout << "Y: \n" << Y << "\n";

    s = ney::apply<int>(ney::operation::mul).vector1(X).value(a);
    s = ney::apply<int>(ney::operation::add).vector1(Y).vector2(X);

    std::cout << "result: \n" << Y << "\n";

    // print time

    ney::clock::get().end().print();

    return 0;
}
