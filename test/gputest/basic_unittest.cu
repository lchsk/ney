#include "hip/hip_runtime.h"
#include "gtest/gtest.h"
#include "../../../include/ney/ney.hpp"

#include <unistd.h>

#define ERROR 0.000001

/*
    Tests of basic operations.
*/

TEST(BasicOperations, Fill)
{
    ney::config.target = ney::GPU;

    ney::vector<int> v = ney::new_vector().size(11);
    ney::status s;

    // Check default settings of a status object
    EXPECT_FALSE(s.success());
    EXPECT_EQ(s.error(), "");

    // fill
    s = ney::fill<int>(v).value(2);

    for (int i = 0; i < 11; i++)
        EXPECT_EQ(v[i], 2);

    EXPECT_TRUE(s.success());
    EXPECT_EQ(s.error(), "");

    // use slicing when filling
    s =  ney::fill<int>(v.from(1).to(5).stride(2)).value(5).time();

    EXPECT_TRUE(s.success());
    EXPECT_EQ(s.error(), "");

    EXPECT_EQ(v[0], 2);
    EXPECT_EQ(v[1], 5);
    EXPECT_EQ(v[2], 2);
    EXPECT_EQ(v[3], 5);
    EXPECT_EQ(v[4], 2);
    EXPECT_EQ(v[5], 2);
    EXPECT_EQ(v[6], 2);

    // it should be possible to call a function without a status object with run() function
    // but without status it is not possible to check if parameters are correct
    ney::fill<int>(v.reset()).value(10).run();

    for (int i = 0; i < 11; i++)
        EXPECT_EQ(v[i], 10);
}

TEST(BasicOperations, Random)
{
    ney::vector<int> v = ney::new_vector().size(5);
    ney::status s;

    // Expect an error

    s = ney::random<int>(v).min(10).max(5);

    EXPECT_FALSE(s.success());
    EXPECT_NE(s.error(), "");

    // That should be ok
    s = ney::random<int>(v).min(5).max(10);

    EXPECT_TRUE(s.success());
    EXPECT_EQ(s.error(), "");

    for (int i = 0; i < 5; i++)
    {
        EXPECT_LE(v[i], 10);
        EXPECT_GE(v[i], 5);
    }
}

TEST(BasicOperations, Count)
{
    ney::vector<int> v = ney::new_vector().size(7);
    ney::status s;

    // Expect an error

    s = ney::fill<int>(v).value(5);

    // Check if vector is initialised correctly

    for (int i = 0; i < 7; i++)
        EXPECT_EQ(v[i], 5);

    // Expect it to work ok

    int count;
    s = ney::count<int>(5).in(v).output(count);

    EXPECT_TRUE(s.success());
    EXPECT_EQ(s.error(), "");

    EXPECT_EQ(count, 7);

    s = ney::count<int>(5).in(v).output(count);

    // Expect 0 as a result

    s = ney::count<int>(6).in(v).output(count);

    EXPECT_TRUE(s.success());
    EXPECT_EQ(s.error(), "");

    EXPECT_EQ(count, 0);

    // Count values in a vector slice

    s = ney::count<int>(5).in(v.stride(2).from(1)).output(count);

    EXPECT_TRUE(s.success());
    EXPECT_EQ(s.error(), "");

    EXPECT_EQ(count, 3);

    // Count floating point values

    ney::vector<double> v2 = ney::new_vector().size(5);

    v2.set(0, 2.5);
    v2.set(1, -923.55);
    v2.set(2, 2.5);
    v2.set(3, 0.1244);
    v2.set(4, -923.55);

    s = ney::count<double>(2.5).in(v2).output(count);

    EXPECT_TRUE(s.success());
    EXPECT_EQ(s.error(), "");

    EXPECT_EQ(count, 2);

    // Expect errors in status object

    s = ney::count<double>(2.5).output(count);

    EXPECT_FALSE(s.success());

    s = ney::count<double>(2.5);

    EXPECT_FALSE(s.success());

    s = ney::count<double>(2.5).in(v2);

    EXPECT_FALSE(s.success());
}
//
TEST(BasicOperations, Replace)
{
    ney::status s;

    ney::vector<int> v1 = ney::new_vector().size(5);
    ney::vector<double> v2 = ney::new_vector().size(5);

    v1.set(0, 1);
    v1.set(1, 10);
    v1.set(2, 50);
    v1.set(3, 10);
    v1.set(4, 50);

    v2.set(0, 0.1);
    v2.set(1, 0.1);
    v2.set(2, 0.3);
    v2.set(3, 5.1);
    v2.set(4, 0.3);

    // Expect errors in status object

    s = ney::replace<double>(2.5);

    EXPECT_FALSE(s.success());

    // TODO: that should also cause an error
    // s = ney::replace<double>(2.5).in(v2);

    // EXPECT_FALSE(s.success());

    s = ney::replace<double>(2.5).with(2.1);

    EXPECT_FALSE(s.success());

    // It should be correct, but no values will be replaced

    s = ney::replace<double>(2.5).with(1.2).in(v2);

    EXPECT_TRUE(s.success());

    int count;
    s = ney::count<double>(2.5).in(v2).output(count);

    EXPECT_EQ(count, 0);

    // That should work

    s = ney::replace<double>(0.3).with(-0.3).in(v2);

    EXPECT_TRUE(s.success());

    s = ney::count<double>(-0.3).in(v2).output(count);

    EXPECT_EQ(count, 2);

    // Replacing integers

    s = ney::replace<int>(10).with(100).in(v1);

    EXPECT_TRUE(s.success());

    s = ney::count<int>(100).in(v1).output(count);

    EXPECT_EQ(count, 2);

    // Replacing doubles with changed precision

    s = ney::replace<double>(0.1).with(0.01).in(v2).precision(0.0001);

    EXPECT_TRUE(s.success());

    s = ney::count<double>(0.01).in(v2).output(count);

    EXPECT_EQ(count, 2);
}
//
// TEST(BasicOperations, Swap)
// {
//     ney::status s;
//
//     ney::vector<double> v1 = ney::new_vector().size(5);
//     ney::vector<double> v2 = ney::new_vector().size(6);
//
//     v1 << 2.1 << 5.2 << 6.6 << -0.01 << 30.5;
//     v2 << 0.2 << 0.5 << 0.2 << 0.67 << 0.23 << 0.21;
//
//     // test swapping values inside a single vector
//
//     s = ney::swap<double>().vector1(v1);
//
//     EXPECT_TRUE(s.success());
//
//     EXPECT_DOUBLE_EQ(v1[0], 6.6);
//     EXPECT_DOUBLE_EQ(v1[1], -0.01);
//     EXPECT_DOUBLE_EQ(v1[2], 2.1);
//     EXPECT_DOUBLE_EQ(v1[3], 5.2);
//     EXPECT_DOUBLE_EQ(v1[4], 30.5);
//
//     s = ney::swap<double>().vector1(v2);
//
//     EXPECT_TRUE(s.success());
//
//     EXPECT_DOUBLE_EQ(v2[0], 0.67);
//     EXPECT_DOUBLE_EQ(v2[1], 0.23);
//     EXPECT_DOUBLE_EQ(v2[2], 0.21);
//     EXPECT_DOUBLE_EQ(v2[3], 0.2);
//     EXPECT_DOUBLE_EQ(v2[4], 0.5);
//     EXPECT_DOUBLE_EQ(v2[5], 0.2);
//
//     // test swapping two vectors
//
//     v1.reset();
//     v2.reset();
//
//     // change size of v1
//
//     v1 = ney::new_vector().size(6);
//
//     EXPECT_EQ(v1.size(), 6);
//     EXPECT_EQ(v1.length(), 6);
//
//     v1 << 0.1 << 0.2 << 0.3 << 0.4 << 0.5 << 0.6;
//     v2 << 1.1 << 1.2 << 1.3 << 1.4 << 1.5 << 1.6;
//
//     s = ney::swap<double>().vector1(v1).vector2(v2);
//
//     EXPECT_DOUBLE_EQ(v1[0], 1.1);
//     EXPECT_DOUBLE_EQ(v1[1], 1.2);
//     EXPECT_DOUBLE_EQ(v1[2], 1.3);
//     EXPECT_DOUBLE_EQ(v1[3], 1.4);
//     EXPECT_DOUBLE_EQ(v1[4], 1.5);
//     EXPECT_DOUBLE_EQ(v1[5], 1.6);
//
//     EXPECT_DOUBLE_EQ(v2[0], 0.1);
//     EXPECT_DOUBLE_EQ(v2[1], 0.2);
//     EXPECT_DOUBLE_EQ(v2[2], 0.3);
//     EXPECT_DOUBLE_EQ(v2[3], 0.4);
//     EXPECT_DOUBLE_EQ(v2[4], 0.5);
//     EXPECT_DOUBLE_EQ(v2[5], 0.6);
//
//     // check swapping with a slice
//
//     v1.reset();
//     v2.reset();
//
//     v1 << 0.1 << 0.2 << 0.3 << 0.4 << 0.5 << 0.6;
//     v2 << 1.1 << 1.2 << 1.3 << 1.4 << 1.5 << 1.6;
//
//     s = ney::swap<double>().vector1(v1.stride(2)).vector2(v2.stride(2));
//
//     EXPECT_DOUBLE_EQ(v1[0], 1.1);
//     EXPECT_DOUBLE_EQ(v1[1], 0.2);
//     EXPECT_DOUBLE_EQ(v1[2], 1.3);
//     EXPECT_DOUBLE_EQ(v1[3], 0.4);
//     EXPECT_DOUBLE_EQ(v1[4], 1.5);
//     EXPECT_DOUBLE_EQ(v1[5], 0.6);
//
//     EXPECT_DOUBLE_EQ(v2[0], 0.1);
//     EXPECT_DOUBLE_EQ(v2[1], 1.2);
//     EXPECT_DOUBLE_EQ(v2[2], 0.3);
//     EXPECT_DOUBLE_EQ(v2[3], 1.4);
//     EXPECT_DOUBLE_EQ(v2[4], 0.5);
//     EXPECT_DOUBLE_EQ(v2[5], 1.6);
//
// }

int main (int argc, char** argv) {
    ::testing::InitGoogleTest(&argc, argv);

    int returnValue;

    returnValue =  RUN_ALL_TESTS();

    return returnValue;
}
