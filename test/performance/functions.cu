#include "hip/hip_runtime.h"
#define USE_MIC FALSE
#include <iostream>
#include "../../include/ney/ney.hpp"

#define TYPE double

int main (int argc, char** argv)
{
    int size;

    if (argv[1] == NULL)
    {
        std::cerr << "usage: " << argv[0] << " <size>" << std::endl;
        return 1;
    }
    else
    {
        size = atoi(argv[1]);
        std::cout << "<size> was set to " << argv[1] << std::endl << std::endl;
    }

    ney::config.target = ney::GPU;

    ney::config.print();

    // start clock

    ney::clock::get().start().precision(7);

    ney::status s;
    TYPE r;
    int c;

    ney::vector<TYPE> x = ney::new_vector().size(size);
    ney::vector<TYPE> y = ney::new_vector().size(size);

    std::cout << "fill\n";
    s = ney::fill<TYPE>(x).value(1).time();
    std::cout << s << "\n";
    std::cout << size * 1e-9 / s.time() << " Gwrites\n\n";

    std::cout << "random\n";
    s = ney::random<TYPE>(x).min(0).max(1).time();
    std::cout << s << "\n";
    std::cout << size * 1e-9 / s.time() << " Gwrites\n\n";

    std::cout << "replace\n";
    s = ney::replace<TYPE>(0.1).with(0.2).in(x);
    std::cout << s << "\n";
    std::cout << size * 1e-9 / s.time() << " Gcomparisons\n\n";

    std::cout << "reduce\n";
    s = ney::reduce<TYPE>(x).use(ney::operation::add).output(r).time();
    std::cout << s << "\n";
    std::cout << size * 1e-9 / s.time() << " Gflops\n\n";

    std::cout << "count\n";
    s = ney::count<TYPE>(0.2).precision(2).in(x).output(c).time();
    std::cout << s << "\n";
    std::cout << size * 1e-9 / s.time() << " Gcomparisons\n\n";

    std::cout << "apply (v1 + v2)\n";
    s = ney::apply<TYPE>(ney::operation::add).vector1(x).vector2(y).time();
    std::cout << s << "\n";
    std::cout << size * 1e-9 / s.time() << " Gflops\n\n";

    std::cout << "sort\n";
    s = ney::sort<TYPE>(y).time();
    std::cout << s << "\n\n";

    std::cout << "unique\n";
    s = ney::unique<TYPE>(x).time();
    std::cout << s << "\n\n";

    // print time

    ney::clock::get().end().print();

    return 0;
}
