#include "hip/hip_runtime.h"
#define USE_MIC FALSE
#include <iostream>
#include "../../include/ney/ney.hpp"

int main (int argc, char** argv)
{
    // int size;
    //
    // if (argv[1] == NULL)
    // {
    //     std::cerr << "usage: " << argv[0] << " <size>" << std::endl;
    //     return 1;
    // }
    // else
    // {
    //     size = atoi(argv[1]);
    // }

    ney::config.target = ney::GPU;

    ney::config.print();

    // start clock

    ney::clock::get().start().precision(7);

    ney::status s;
    ney::vector<int> x1 = ney::new_vector().size(1000);

    s = ney::random<int>(x1).min(0).max(100).time();
    std::cout << s << "\n";

    int r;
    s = ney::reduce<int>(x1).use(ney::operation::add).output(r).time();
    std::cout << s << "\n";

    ney::vector<int> x2 = ney::new_vector().size(100000);

    s = ney::random<int>(x2).min(0).max(100).time();
    std::cout << s << "\n";

    s = ney::reduce<int>(x2).use(ney::operation::add).output(r).time();
    std::cout << s << "\n";

    ney::vector<int> x3 = ney::new_vector().size(1000000);

    s = ney::random<int>(x3).min(0).max(100).time();
    std::cout << s << "\n";

    s = ney::reduce<int>(x3).use(ney::operation::add).output(r).time();
    std::cout << s << "\n";

    // print time

    ney::clock::get().end().print();

    return 0;
}
